#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

vector3 *hVel, *dhVel;
vector3 *hPos, *dhPos;
double *mass;

void initHostMemory(int numObjects){
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

void freeHostMemory(){
	free(hVel);
	free(hPos);
	free(mass);
}

void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

void randomFill(int start, int count){
	int i, j = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

int main(int argc, char **argv){
	clock_t t0=clock();
	int t_now;
	srand(1234);
	initHostMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	#ifdef DEBUG
	printSystem(stdout);
	#endif
	for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
		compute();
	}
	clock_t t1=clock()-t0;
	#ifdef DEBUG
	printSystem(stdout);
	#endif
	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
}
